#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

__global__
void check(char *x, int n, bool* z)
{
  char str [] ="em`hzugbst@oDy`nomdPeTHNS``sbihuddsvqf|"; 
  // s = flag{thatsAnExampleOfSIMT_architecture} -> 
  /* 
  k = ''
  for i in range(len(s)):
	add = (i % 2) * 2 - 1
	k += chr(ord(s[i]) + add)
	*/
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int additive = (i & 1) * 2 - 1;
  if (i < n)
  {
	  z[i] = true;
	  if (str[i] - additive != x[i])
		  z[i] = false;
  }
}

int main(void)
{
  char *d_s;
  bool *f;
  
  std::string s;
  std::cin >> s;
  
  hipMalloc(&d_s, s.length() * sizeof(char)); 
  hipMalloc(&f, s.length() * sizeof(bool));


  hipMemcpy(d_s, s.c_str(), s.length()*sizeof(char), hipMemcpyHostToDevice);

  check<<<1, 1024>>>(d_s, s.length(), f);
  
  thrust::device_ptr<bool> dp = thrust::device_pointer_cast(f);
  thrust::device_ptr<bool> pos = thrust::min_element(thrust::device, dp, dp + s.length());
  


  unsigned int pos_index = thrust::distance(dp, pos);
  bool min_val;
  hipMemcpy(&min_val, &f[pos_index], sizeof(bool), hipMemcpyDeviceToHost);
  
  
  std::cout << min_val << "\n";

  hipFree(d_s);
  hipFree(f);
}